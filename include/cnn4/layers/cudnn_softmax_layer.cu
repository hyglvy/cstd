
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN


#include "thrust/device_vector.h"

#include "caffe/layers/cudnn_softmax_layer.hpp"

namespace {

template <typename Dtype>
void CuDNNSoftmaxLayer::Forward(GPUContext* context, const vector<Blob*>& bottom,
    const vector<Blob*>& top) {
  const Dtype* bottom_data = bottom[0]->data();
  Dtype* top_data = top[0]->mdata();
  CUDNN_CHECK(cudnnSoftmaxForward(handle_, CUDNN_SOFTMAX_ACCURATE,
        CUDNN_SOFTMAX_MODE_CHANNEL,
        cudnn::dataType<Dtype>::one,
        bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        top_desc_, top_data));
}

template <typename Dtype>
void CuDNNSoftmaxLayer::Backward(GPUContext* context, const vector<Blob*>& top,
    const vector<Blob*>& bottom) {
  if (bottom[0]->propagate_down_) {
    const Dtype* top_data = top[0]->data();
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* bottom_data = bottom[0]->data();
    Dtype* bottom_diff = bottom[0]->gpu_mdiff();

    CUDNN_CHECK(cudnnSoftmaxBackward(handle_, CUDNN_SOFTMAX_ACCURATE,
          CUDNN_SOFTMAX_MODE_CHANNEL,
          cudnn::dataType<Dtype>::one,
          top_desc_, top_data, top_desc_, top_diff,
          cudnn::dataType<Dtype>::zero,
          bottom_desc_, bottom_diff));
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNSoftmaxLayer);

}  // namespace
#endif
