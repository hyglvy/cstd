#include "hip/hip_runtime.h"


#include "caffe/filler.hpp"
#include "caffe/layers/bias_layer.hpp"


namespace {

template <typename Dtype>
__global__ void BiasForward(const int n, const Dtype* in,
    const Dtype* bias, const int bias_dim, const int inner_dim,
    Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    const int bias_index = (index / inner_dim) % bias_dim;
    out[index] = in[index] + bias[bias_index];
  }
}

template <typename Dtype>
void BiasLayer::Forward(GPUContext* context, const vector<Blob*>& bottom,
      const vector<Blob*>& top) {
  const int count = top[0]->count();
  const Dtype* bottom_data = bottom[0]->data();
  const Dtype* bias_data =
      ((bottom.size() > 1) ? bottom[1] : this->blobs_[0].get())->data();
  Dtype* top_data = top[0]->mdata();
  BiasForward<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, bias_data, bias_dim_, inner_dim_, top_data);
}

template <typename Dtype>
void BiasLayer::Backward(GPUContext* context, const vector<Blob*>& top,
      const vector<Blob*>& bottom) {
  if (bottom[0]->propagate_down_ && bottom[0] != top[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->gpu_mdiff();
    caffe_copy(bottom[0]->count(), top_diff, bottom_diff);
  }
  // in-place, we don't need to do anything with the data diff
  const bool bias_param = (bottom.size() == 1);
  if ((!bias_param && bottom[1]->propagate_down_) ||
      (bias_param && this->blobs_[0]->propagate_down_)) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bias_diff = (bias_param ? this->blobs_[0].get() : bottom[1])
        ->gpu_mdiff();
    bool accum = bias_param;
    for (int n = 0; n < outer_dim_; ++n) {
      caffe_gpu_gemv(CblasNoTrans, bias_dim_, inner_dim_, Dtype(1),
          top_diff, bias_multiplier_.data(), Dtype(accum), bias_diff);
      top_diff += dim_;
      accum = true;
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(BiasLayer);

}  // namespace
