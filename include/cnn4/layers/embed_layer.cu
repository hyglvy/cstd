#include "hip/hip_runtime.h"


#include "caffe/filler.hpp"
#include "caffe/layers/embed_layer.hpp"
#include "caffe/util/gpu_util.cuh"


namespace {

template <typename Dtype>
__global__ void EmbedForward(const int nthreads, const Dtype* bottom_data,
    const Dtype* weight, const int M, const int N, const int K,
    Dtype* top_data) {
  CUDA_KERNEL_LOOP(top_index, nthreads) {
    const int n = top_index / N;
    const int d = top_index % N;
    const int index = static_cast<int>(bottom_data[n]);
    const int weight_index = index * N + d;
    top_data[top_index] = weight[weight_index];
  }
}

template <typename Dtype>
__global__ void EmbedBackward(const int nthreads, const Dtype* bottom_data,
    const Dtype* top_diff, const int M, const int N, const int K,
    Dtype* weight_diff);

template <typename Dtype>
__global__ void EmbedBackward(const int nthreads, const Dtype* bottom_data,
    const Dtype* top_diff, const int M, const int N, const int K,
    Dtype* weight_diff) {
  CUDA_KERNEL_LOOP(top_index, nthreads) {
    const int n = top_index / N;
    const int d = top_index % N;
    const int index = static_cast<int>(bottom_data[n]);
    const int weight_index = index * N + d;
    caffe_gpu_atomic_add(top_diff[top_index], weight_diff + weight_index);
  }
}

template <typename Dtype>
void EmbedLayer::Forward(GPUContext* context, const vector<Blob*>& bottom,
    const vector<Blob*>& top) {
  const Dtype* bottom_data = bottom[0]->data();
  Dtype* top_data = top[0]->mdata();
  const Dtype* weight = this->blobs_[0]->data();
  const int count = top[0]->count();
  EmbedForward<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, weight, M_, N_, K_, top_data);
  if (bias_term_) {
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, N_, 1, Dtype(1),
        bias_multiplier_.data(),
        this->blobs_[1]->data(), Dtype(1), top_data);
  }
}

template <typename Dtype>
void EmbedLayer::Backward(GPUContext* context, const vector<Blob*>& top,
    const vector<Blob*>& bottom) {
  CHECK(!bottom[0]->propagate_down_) << "Can't backpropagate to EmbedLayer input.";
  if (this->blobs_[0]->propagate_down_) {
    const int top_count = top[0]->count();
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* bottom_data = bottom[0]->data();
    Dtype* weight_diff = this->blobs_[0]->gpu_mdiff();
    EmbedBackward<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(top_count), CAFFE_CUDA_NUM_THREADS>>>(
        top_count, bottom_data, top_diff, M_, N_, K_, weight_diff);
  }
  if (bias_term_ && this->blobs_[1]->propagate_down_) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bias_diff = this->blobs_[1]->gpu_mdiff();
    caffe_gpu_gemv<Dtype>(CblasTrans, M_, N_, Dtype(1), top_diff,
        bias_multiplier_.data(), Dtype(1), bias_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(EmbedLayer);

}  // namespace
