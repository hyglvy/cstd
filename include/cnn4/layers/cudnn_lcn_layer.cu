
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN


#include "caffe/layers/cudnn_lcn_layer.hpp"

namespace {

template <typename Dtype>
void CuDNNLCNLayer::Forward(GPUContext* context, const vector<Blob*>& bottom,
    const vector<Blob*>& top) {
  const Dtype* bottom_data = bottom[0]->data();
  Dtype* top_data = top[0]->mutable_data();

  CUDNN_CHECK(cudnnDivisiveNormalizationForward(
        handle_, norm_desc_, CUDNN_DIVNORM_PRECOMPUTED_MEANS,
        cudnn::dataType<Dtype>::one,
        bottom_desc_, bottom_data,
        NULL,  // srcMeansData
        this->tempData1, this->tempData2,
        cudnn::dataType<Dtype>::zero,
        top_desc_, top_data) );
}

template <typename Dtype>
void CuDNNLCNLayer::Backward(GPUContext* context, const vector<Blob*>& top,
    const vector<bool>& propagate_down, const vector<Blob*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->data();
  const Dtype* bottom_data = bottom[0]->data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

  CUDNN_CHECK(cudnnDivisiveNormalizationBackward(
        handle_, norm_desc_, CUDNN_DIVNORM_PRECOMPUTED_MEANS,
        cudnn::dataType<Dtype>::one,
        bottom_desc_, bottom_data,
        NULL, top_diff,  // NULL - srcMeansData
        this->tempData1, this->tempData2,
        cudnn::dataType<Dtype>::zero,
        bottom_desc_, bottom_diff,
        NULL) );
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNLCNLayer);

}  // namespace
#endif
