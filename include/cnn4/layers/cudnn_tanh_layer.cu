
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN


#include "caffe/layers/cudnn_tanh_layer.hpp"

namespace {

template <typename Dtype>
void CuDNNTanHLayer::Forward(GPUContext* context, const vector<Blob*>& bottom,
    const vector<Blob*>& top) {
  const Dtype* bottom_data = bottom[0]->data();
  Dtype* top_data = top[0]->mdata();
#if CUDNN_VERSION_MIN(5, 0, 0)
  CUDNN_CHECK(cudnnActivationForward(this->handle_,
        activ_desc_,
        cudnn::dataType<Dtype>::one,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->top_desc_, top_data));
#else
  CUDNN_CHECK(cudnnActivationForward_v4(this->handle_,
        activ_desc_,
        cudnn::dataType<Dtype>::one,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->top_desc_, top_data));
#endif
}

template <typename Dtype>
void CuDNNTanHLayer::Backward(GPUContext* context, const vector<Blob*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob*>& bottom) {
  if (!bottom[0]->propagate_down_) {
    return;
  }

  const Dtype* top_data = top[0]->data();
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* bottom_data = bottom[0]->data();
  Dtype* bottom_diff = bottom[0]->gpu_mdiff();

#if CUDNN_VERSION_MIN(5, 0, 0)
  CUDNN_CHECK(cudnnActivationBackward(this->handle_,
        activ_desc_,
        cudnn::dataType<Dtype>::one,
        this->top_desc_, top_data, this->top_desc_, top_diff,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->bottom_desc_, bottom_diff));
#else
  CUDNN_CHECK(cudnnActivationBackward_v4(this->handle_,
        activ_desc_,
        cudnn::dataType<Dtype>::one,
        this->top_desc_, top_data, this->top_desc_, top_diff,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->bottom_desc_, bottom_diff));
#endif
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNTanHLayer);

}  // namespace
#endif
