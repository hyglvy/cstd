
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN


#include "caffe/layers/cudnn_lrn_layer.hpp"

namespace {

template <typename Dtype>
void CuDNNLRNLayer::Forward(GPUContext* context, const vector<Blob*>& bottom,
    const vector<Blob*>& top) {
  const Dtype* bottom_data = bottom[0]->data();
  Dtype* top_data = top[0]->mutable_data();

  CUDNN_CHECK(cudnnLRNCrossChannelForward(
        handle_, norm_desc_, CUDNN_LRN_CROSS_CHANNEL_DIM1,
        cudnn::dataType<Dtype>::one,
        bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        top_desc_, top_data) );
}

template <typename Dtype>
void CuDNNLRNLayer::Backward(GPUContext* context, const vector<Blob*>& top,
    const vector<bool>& propagate_down, const vector<Blob*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->data();
  const Dtype* bottom_data = bottom[0]->data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

  CUDNN_CHECK(cudnnLRNCrossChannelBackward(
        handle_, norm_desc_, CUDNN_LRN_CROSS_CHANNEL_DIM1,
        cudnn::dataType<Dtype>::one,
        top_desc_, top_data,
        top_desc_, top_diff,
        bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        bottom_desc_, bottom_diff) );
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNLRNLayer);

};  // namespace

#endif
