
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN


#include "caffe/layers/cudnn_lrn_layer.hpp"

namespace {

template <typename Dtype>
void CuDNNLRNLayer<Dtype>::Forward(GPUContext* context, const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->data<Context>();
  Dtype* top_data = top[0]->mutable_data<Context>();

  CUDNN_CHECK(cudnnLRNCrossChannelForward(
        handle_, norm_desc_, CUDNN_LRN_CROSS_CHANNEL_DIM1,
        cudnn::dataType<Dtype>::one,
        bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        top_desc_, top_data) );
}

template <typename Dtype>
void CuDNNLRNLayer<Dtype>::Backward(GPUContext* context, const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->data<Context>();
  const Dtype* bottom_data = bottom[0]->data<Context>();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

  CUDNN_CHECK(cudnnLRNCrossChannelBackward(
        handle_, norm_desc_, CUDNN_LRN_CROSS_CHANNEL_DIM1,
        cudnn::dataType<Dtype>::one,
        top_desc_, top_data,
        top_desc_, top_diff,
        bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        bottom_desc_, bottom_diff) );
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNLRNLayer);

};  // namespace

#endif
