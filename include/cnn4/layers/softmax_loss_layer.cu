#include "hip/hip_runtime.h"

#ifndef Dtype

#include <algorithm>
#include <cfloat>



template <typename Dtype>
__global__ void SoftmaxLossForwardGPU(const int nthreads,
          const Dtype* prob_data, const Dtype* label, Dtype* loss,
          const int num, const int dim, const int spatial_dim,
          const bool has_ignore_label_, const int ignore_label_,
          Dtype* counts) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int label_value = static_cast<int>(label[n * spatial_dim + s]);
    if (has_ignore_label_ && label_value == ignore_label_) {
      loss[index] = 0;
      counts[index] = 0;
    } else {
      loss[index] = -log(max(prob_data[n * dim + label_value * spatial_dim + s],
                      Dtype(FLT_MIN)));
      counts[index] = 1;
    }
  }
}

template <typename Dtype>
__global__ void SoftmaxLossBackwardGPU(const int nthreads, const Dtype* top,
          const Dtype* label, Dtype* bottom_diff, const int num, const int dim,
          const int spatial_dim, const bool has_ignore_label_,
          const int ignore_label_, Dtype* counts) {
  const int channels = dim / spatial_dim;

  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int label_value = static_cast<int>(label[n * spatial_dim + s]);

    if (has_ignore_label_ && label_value == ignore_label_) {
      for (int c = 0; c < channels; ++c) {
        bottom_diff[n * dim + c * spatial_dim + s] = 0;
      }
      counts[index] = 0;
    } else {
      bottom_diff[n * dim + label_value * spatial_dim + s] -= 1;
      counts[index] = 1;
    }
  }
}

#else

template <>
int softmaxloss_forward(const Dtype* prob_data, const Dtype* label,
  const int outer_num_, const int dim, const int inner_num_,
  const bool has_ignore_label_, const int ignore_label_, Dtype* out_loss) {
  const int nthreads = outer_num_ * inner_num_;
  BufData<Dtype, GPUContext> counts_buf(nthreads);
  BufData<Dtype, GPUContext> loss_buf(nthreads);
  Dtype* counts = counts_buf.mutable_get();
  Dtype* loss_data = loss_buf.mutable_get();
  SoftmaxLossForwardGPU<Dtype> << <CAFFE_GET_BLOCKS(nthreads),
    CAFFE_CUDA_NUM_THREADS >> >(nthreads, prob_data, label, loss_data,
      outer_num_, dim, inner_num_, has_ignore_label_, ignore_label_, counts);
  caffe_asum(nthreads, loss_data, out_loss);
  Dtype valid_count = 0;
  // Only launch another CUDA kernel if we actually need the count of valid
  // outputs.
  caffe_asum(nthreads, counts, &valid_count);
  valid_count -= 1;
  return (int)valid_count;
}

template <>
int softmaxloss_backward(_CONTEXT,const Dtype* top_data,
  const Dtype* label, Dtype* bottom_diff, const int outer_num_, const int dim,
  const int inner_num_, const bool has_ignore_label_,
  const int ignore_label_) {
  const int nthreads = outer_num_ * inner_num_;
  BufData<Dtype, GPUContext> counts_buf(nthreads);
  Dtype* counts = counts_buf.mutable_get();

  Dtype valid_count = 0;
  SoftmaxLossBackwardGPU<Dtype> << <CAFFE_GET_BLOCKS(nthreads),
    CAFFE_CUDA_NUM_THREADS >> >(nthreads, top_data, label, bottom_diff,
      outer_num_, dim, inner_num_, has_ignore_label_, ignore_label_, counts);
  caffe_asum(nthreads, counts, &valid_count);
  valid_count -= 1;
  return (int)valid_count;
}

#endif
