#include "hip/hip_runtime.h"

#include <algorithm>
#include <cfloat>

__global__ void FUN(SoftmaxLossForwardGPU)(const int nthreads,
          const Dtype* prob_data, const Dtype* label, Dtype* loss,
          const int num, const int dim, const int spatial_dim,
          const bool has_ignore_label_, const int ignore_label_,
          Dtype* counts) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int label_value = static_cast<int>(label[n * spatial_dim + s]);
    if (has_ignore_label_ && label_value == ignore_label_) {
      loss[index] = 0;
      counts[index] = 0;
    } else {
      loss[index] = -log(max(prob_data[n * dim + label_value * spatial_dim + s],
                      Dtype(FLT_MIN)));
      counts[index] = 1;
    }
  }
}

__global__ void FUN(SoftmaxLossBackwardGPU)(const int nthreads, const Dtype* top,
          const Dtype* label, Dtype* bottom_diff, const int num, const int dim,
          const int spatial_dim, const bool has_ignore_label_,
          const int ignore_label_, Dtype* counts) {
  const int channels = dim / spatial_dim;

  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int label_value = static_cast<int>(label[n * spatial_dim + s]);

    if (has_ignore_label_ && label_value == ignore_label_) {
      for (int c = 0; c < channels; ++c) {
        bottom_diff[n * dim + c * spatial_dim + s] = 0;
      }
      counts[index] = 0;
    } else {
      bottom_diff[n * dim + label_value * spatial_dim + s] -= 1;
      counts[index] = 1;
    }
  }
}


int FUN(softmaxloss_forward)(const Dtype* prob_data, const Dtype* label,
  const int outer_num_, const int dim, const int inner_num_,
  const bool has_ignore_label_, const int ignore_label_, Dtype* out_loss) {
  const int nthreads = outer_num_ * inner_num_;
  BufData<Dtype> counts_buf(GPU, nthreads);
  BufData<Dtype> loss_buf(GPU, nthreads);
  Dtype* counts = counts_buf.get();
  Dtype* loss_data = loss_buf.get();
  FUN(SoftmaxLossForwardGPU) << <CAFFE_GET_BLOCKS(nthreads),
    CAFFE_CUDA_NUM_THREADS >> >(nthreads, prob_data, label, loss_data,
      outer_num_, dim, inner_num_, has_ignore_label_, ignore_label_, counts);
  FUN(caffe_asum2)(nthreads, loss_data, out_loss);
  Dtype valid_count = 0;
  // Only launch another CUDA kernel if we actually need the count of valid
  // outputs.
  FUN(caffe_asum2)(nthreads, counts, &valid_count);
  valid_count -= 1;
  return (int)valid_count;
}

int FUN(softmaxloss_backward)(const Dtype* top_data,
  const Dtype* label, Dtype* bottom_diff, const int outer_num_, const int dim,
  const int inner_num_, const bool has_ignore_label_,
  const int ignore_label_) {
  const int nthreads = outer_num_ * inner_num_;
  BufData<Dtype> counts_buf(GPU, nthreads);
  Dtype* counts = counts_buf.get();

  Dtype valid_count = 0;
  FUN(SoftmaxLossBackwardGPU) << <CAFFE_GET_BLOCKS(nthreads),
    CAFFE_CUDA_NUM_THREADS >> >(nthreads, top_data, label, bottom_diff,
      outer_num_, dim, inner_num_, has_ignore_label_, ignore_label_, counts);
  FUN(caffe_asum2)(nthreads, counts, &valid_count);
  valid_count -= 1;
  return (int)valid_count;
}
