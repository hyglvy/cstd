
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN


#include "caffe/layers/cudnn_pooling_layer.hpp"

namespace {

template <typename Dtype>
void CuDNNPoolingLayer::Forward_(GPUContext* context, const vector<Blob*>& bottom,
    const vector<Blob*>& top) {
  const Dtype* bottom_data = bottom[0]->data();
  Dtype* top_data = top[0]->mdata();
  CUDNN_CHECK(cudnnPoolingForward(handle_, pooling_desc_,
        cudnn::dataType<Dtype>::one,
        bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        top_desc_, top_data));
}

template <typename Dtype>
void CuDNNPoolingLayer::Backward_(GPUContext* context, const vector<Blob*>& top,
    const vector<Blob*>& bottom) {
  if (!bottom[0]->propagate_down_) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->data();
  const Dtype* bottom_data = bottom[0]->data();
  Dtype* bottom_diff = bottom[0]->gpu_mdiff();
  CUDNN_CHECK(cudnnPoolingBackward(handle_, pooling_desc_,
        cudnn::dataType<Dtype>::one,
        top_desc_, top_data, top_desc_, top_diff,
        bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        bottom_desc_, bottom_diff));
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNPoolingLayer);

}  // namespace
#endif
