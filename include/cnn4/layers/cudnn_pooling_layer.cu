
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN


#include "caffe/layers/cudnn_pooling_layer.hpp"

namespace {

template <typename Dtype>
void CuDNNPoolingLayer::Forward(GPUContext* context, const vector<Blob*>& bottom,
    const vector<Blob*>& top) {
  const Dtype* bottom_data = bottom[0]->data();
  Dtype* top_data = top[0]->mutable_data();
  CUDNN_CHECK(cudnnPoolingForward(handle_, pooling_desc_,
        cudnn::dataType<Dtype>::one,
        bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        top_desc_, top_data));
}

template <typename Dtype>
void CuDNNPoolingLayer::Backward(GPUContext* context, const vector<Blob*>& top,
    const vector<bool>& propagate_down, const vector<Blob*>& bottom) {
  if (!bottom[0]->propagate_down_) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->data();
  const Dtype* bottom_data = bottom[0]->data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  CUDNN_CHECK(cudnnPoolingBackward(handle_, pooling_desc_,
        cudnn::dataType<Dtype>::one,
        top_desc_, top_data, top_desc_, top_diff,
        bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        bottom_desc_, bottom_diff));
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNPoolingLayer);

}  // namespace
#endif
