
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN


#include "caffe/layers/cudnn_relu_layer.hpp"

namespace {

template <typename Dtype>
void CuDNNReLULayer::Forward_(GPUContext* context, const vector<Blob*>& bottom,
    const vector<Blob*>& top) {
  // Fallback to standard Caffe for leaky ReLU.
  if (ReLULayer::param_->relu_param().negative_slope() != 0) {
    return ReLULayer::Forward_gpu(bottom, top);
  }

  const Dtype* bottom_data = bottom[0]->data();
  Dtype* top_data = top[0]->mdata();
#if CUDNN_VERSION_MIN(5, 0, 0)
  CUDNN_CHECK(cudnnActivationForward(this->handle_,
        activ_desc_,
        cudnn::dataType<Dtype>::one,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->top_desc_, top_data));
#else
  CUDNN_CHECK(cudnnActivationForward_v4(this->handle_,
        activ_desc_,
        cudnn::dataType<Dtype>::one,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->top_desc_, top_data));
#endif
}

template <typename Dtype>
void CuDNNReLULayer::Backward_(GPUContext* context, const vector<Blob*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob*>& bottom) {
  if (!bottom[0]->propagate_down_) {
    return;
  }

  // Fallback to standard Caffe for leaky ReLU.
  if (ReLULayer::param_->relu_param().negative_slope() != 0) {
    return ReLULayer::Backward_gpu(top, propagate_down, bottom);
  }

  const Dtype* top_data = top[0]->data();
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* bottom_data = bottom[0]->data();
  Dtype* bottom_diff = bottom[0]->gpu_mdiff();
#if CUDNN_VERSION_MIN(5, 0, 0)
  CUDNN_CHECK(cudnnActivationBackward(this->handle_,
        activ_desc_,
        cudnn::dataType<Dtype>::one,
        this->top_desc_, top_data, this->top_desc_, top_diff,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->bottom_desc_, bottom_diff));
#else
  CUDNN_CHECK(cudnnActivationBackward_v4(this->handle_,
        activ_desc_,
        cudnn::dataType<Dtype>::one,
        this->top_desc_, top_data, this->top_desc_, top_diff,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->bottom_desc_, bottom_diff));
#endif
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNReLULayer);

}  // namespace
#endif
