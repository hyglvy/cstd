#include "hip/hip_runtime.h"
#include <algorithm>


#include "caffe/layers/contrastive_loss_layer.hpp"


namespace {

template <typename Dtype>
void ContrastiveLossLayer::Forward_gpu(
    const vector<Blob*>& bottom, const vector<Blob*>& top) {
  const int count = bottom[0]->count();
  caffe_gpu_sub(
      count,
      bottom[0]->data(),  // a
      bottom[1]->data(),  // b
      diff_.mdata());  // a_i-b_i
  caffe_gpu_powx(
      count,
      diff_.mdata(),  // a_i-b_i
      Dtype(2),
      diff_sq_.mdata());  // (a_i-b_i)^2
  caffe_gpu_gemv(
      CblasNoTrans,
      bottom[0]->num(),
      bottom[0]->channels(),
      Dtype(1.0),
      diff_sq_.data(),  // (a_i-b_i)^2
      summer_vec_.data(),
      Dtype(0.0),
      dist_sq_.mdata());  // \Sum (a_i-b_i)^2
  Dtype margin = this->param_->contrastive_loss_param().margin();
  bool legacy_version =
      this->param_->contrastive_loss_param().legacy_version();
  Dtype loss(0.0);
  for (int i = 0; i < bottom[0]->num(); ++i) {
    if (static_cast<int>(bottom[2]->data()[i])) {  // similar pairs
      loss += dist_sq_.data()[i];
    } else {  // dissimilar pairs
      if (legacy_version) {
        loss += std::max(margin - dist_sq_.data()[i], Dtype(0.0));
      } else {
        Dtype dist = std::max(margin - sqrt(dist_sq_.data()[i]),
                              Dtype(0.0));
        loss += dist*dist;
      }
    }
  }
  loss = loss / static_cast<Dtype>(bottom[0]->num()) / Dtype(2);
  top[0]->mdata()[0] = loss;
}

template <typename Dtype>
__global__ void CLLBackward(const int count, const int channels,
    const Dtype margin, const bool legacy_version, const Dtype alpha,
    const Dtype* y, const Dtype* diff, const Dtype* dist_sq,
    Dtype *bottom_diff) {
  CUDA_KERNEL_LOOP(i, count) {
    int n = i / channels;  // the num index, to access y and dist_sq
    if (static_cast<int>(y[n])) {  // similar pairs
      bottom_diff[i] = alpha * diff[i];
    } else {  // dissimilar pairs
      Dtype mdist(0.0);
      Dtype beta(0.0);
      if (legacy_version) {
        mdist = (margin - dist_sq[n]);
        beta = -alpha;
      } else {
        Dtype dist = sqrt(dist_sq[n]);
        mdist = (margin - dist);
        beta = -alpha * mdist / (dist + Dtype(1e-4)) * diff[i];
      }
      if (mdist > 0.0) {
        bottom_diff[i] = beta;
      } else {
        bottom_diff[i] = 0;
      }
    }
  }
}

template <typename Dtype>
void ContrastiveLossLayer::Backward_(GPUContext* context, const vector<Blob*>& top,
    const vector<Blob*>& bottom) {
  for (int i = 0; i < 2; ++i) {
    if (bottom[i]->propagate_down_) {
      const int count = bottom[0]->count();
      const int channels = bottom[0]->channels();
      Dtype margin = this->param_->contrastive_loss_param().margin();
      const bool legacy_version =
          this->param_->contrastive_loss_param().legacy_version();
      const Dtype sign = Dtype((i == 0) ? 1 : -1);
      const Dtype alpha = sign * top[0]->diff()[0] /
          static_cast<Dtype>(bottom[0]->num());
      // NOLINT_NEXT_LINE(whitespace/operators)
      CLLBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, channels, margin, legacy_version, alpha,
          bottom[2]->data(),  // pair similarity 0 or 1
          diff_.data(),  // the cached eltwise difference between a and b
          dist_sq_.data(),  // the cached square distance between a and b
          bottom[i]->gpu_mdiff());
      CUDA_POST_KERNEL_CHECK;
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ContrastiveLossLayer);

}  // namespace
