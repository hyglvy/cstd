#include "hip/hip_runtime.h"

__global__ void FUN(ScaleForward)(const int n, const Dtype* in,
    const Dtype* scale, const int scale_dim, const int inner_dim,
    Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    const int scale_index = (index / inner_dim) % scale_dim;
    out[index] = in[index] * scale[scale_index];
  }
}

void FUN(scale_forward)(const int n, const Dtype* in, const Dtype* scale, const int scale_dim, const int inner_dim, Dtype* out) {
  FUN(ScaleForward) << <CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS >> >( n, in, scale, scale_dim, inner_dim, out);
}
__global__ void FUN(ScaleBiasForward)(const int n, const Dtype* in,
    const Dtype* scale, const Dtype* bias,
    const int scale_dim, const int inner_dim, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    const int scale_index = (index / inner_dim) % scale_dim;
    out[index] = in[index] * scale[scale_index] + bias[scale_index];
  }
}
void FUN(scalebias_forward)(const int n, const Dtype* in,const Dtype* scale, const Dtype* bias, const int scale_dim, const int inner_dim, Dtype* out) {
  FUN(ScaleBiasForward) << <CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS >> >(n, in, scale, bias, scale_dim, inner_dim, out);
}
