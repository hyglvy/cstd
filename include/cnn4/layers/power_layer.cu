#include "hip/hip_runtime.h"


#include "caffe/layers/power_layer.hpp"


namespace {

template <typename Dtype>
void PowerLayer::Forward_(GPUContext* context, const vector<Blob*>& bottom,
    const vector<Blob*>& top) {
  Dtype* top_data = top[0]->mdata();
  const int count = bottom[0]->count();
  // Special case where we can ignore the input: scale or power is 0.
  if (diff_scale_ == Dtype(0)) {
    Dtype value = (power_ == 0) ? Dtype(1) : pow(shift_, power_);
    caffe_gpu_set(count, value, top_data);
    return;
  }
  const Dtype* bottom_data = bottom[0]->data();
  caffe_copy(count, bottom_data, top_data);
  if (scale_ != Dtype(1)) {
    caffe_gpu_scal(count, scale_, top_data);
  }
  if (shift_ != Dtype(0)) {
    caffe_gpu_add_scalar(count, shift_, top_data);
  }
  if (power_ != Dtype(1)) {
    caffe_gpu_powx(count, top_data, power_, top_data);
  }
}

template <typename Dtype>
void PowerLayer::Backward_(GPUContext* context, const vector<Blob*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob*>& bottom) {
  if (bottom[0]->propagate_down_) {
    Dtype* bottom_diff = bottom[0]->gpu_mdiff();
    const int count = bottom[0]->count();
    const Dtype* top_diff = top[0]->gpu_diff();
    if (diff_scale_ == Dtype(0) || power_ == Dtype(1)) {
      caffe_gpu_set(count, diff_scale_, bottom_diff);
    } else {
      const Dtype* bottom_data = bottom[0]->data();
      // Compute dy/dx = scale * power * (shift + scale * x)^(power - 1)
      //               = diff_scale * y / (shift + scale * x)
      if (power_ == Dtype(2)) {
        // Special case for y = (shift + scale * x)^2
        //     -> dy/dx = 2 * scale * (shift + scale * x)
        //              = diff_scale * shift + diff_scale * scale * x
        caffe_gpu_axpby(count, diff_scale_ * scale_, bottom_data,
            Dtype(0), bottom_diff);
        if (shift_ != Dtype(0)) {
          caffe_gpu_add_scalar(count, diff_scale_ * shift_, bottom_diff);
        }
      } else if (shift_ == Dtype(0)) {
        // Special case for y = (scale * x)^power
        //     -> dy/dx = scale * power * (scale * x)^(power - 1)
        //              = scale * power * (scale * x)^power * (scale * x)^(-1)
        //              = power * y / x
        const Dtype* top_data = top[0]->data();
        caffe_gpu_div(count, top_data, bottom_data, bottom_diff);
        caffe_gpu_scal(count, power_, bottom_diff);
      } else {
        caffe_copy(count, bottom_data, bottom_diff);
        if (scale_ != Dtype(1)) {
          caffe_gpu_scal(count, scale_, bottom_diff);
        }
        if (shift_ != Dtype(0)) {
          caffe_gpu_add_scalar(count, shift_, bottom_diff);
        }
        const Dtype* top_data = top[0]->data();
        caffe_gpu_div<Dtype>(count, top_data, bottom_diff, bottom_diff);
        if (diff_scale_ != Dtype(1)) {
          caffe_gpu_scal(count, diff_scale_, bottom_diff);
        }
      }
    }
    caffe_gpu_mul(count, top_diff, bottom_diff, bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(PowerLayer);


}  // namespace
