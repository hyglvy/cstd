#include "hip/hip_runtime.h"
#include "caffe/layers/transpose_layer.hpp"

namespace {

template <typename Dtype>
__global__ void transpose_gpu(const int nthreads, const Dtype* from_data, Dtype* to_data, 
	const int* from_counts, const int* to_counts, const int* map, const int num_axes, int* buf) {
  CUDA_KERNEL_LOOP(index, nthreads) {
  	int* from_inds=buf + index * num_axes;

  	int from_index = index, to_index = 0;
  	for(int i = 0; i < num_axes; i++) {
		from_inds[i] = from_index / from_counts[i];
		from_index = from_index % from_counts[i];
	}
	for(int i = 0; i < num_axes; i++) {
		to_index += from_inds[map[i]] * to_counts[i];
	}

	*(to_data + to_index) = *(from_data + index);
  }
}

template <typename Dtype>
void TransposeLayer::Forward(GPUContext* context, const vector<Blob*>& bottom, 
		const vector<Blob*>& top) {
	const int nthreads = bottom[0]->count();

	transpose_gpu<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
        nthreads, bottom[0]->data(), top[0]->mdata(), 
        bottom_counts_.data(), top_counts_.data(), forward_map_.data(), 
          (int)bottom[0]->shape().size(), buf_.mdata());
}

template <typename Dtype>
void TransposeLayer::Backward(GPUContext* context, const vector<Blob*>& top,
    	const vector<Blob*>& bottom) {
	if (!bottom[0]->propagate_down_) {
		return;
	}
	const int nthreads = bottom[0]->count();

	transpose_gpu<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
        nthreads, top[0]->gpu_diff(), bottom[0]->gpu_mdiff(), 
        top_counts_.data(), bottom_counts_.data(), backward_map_.data(), 
        (int)bottom[0]->shape().size(), buf_.mdata());
}

INSTANTIATE_LAYER_GPU_FUNCS(TransposeLayer);

}  // namespace
