#include "hip/hip_runtime.h"
#include "caffe/layers/transpose_layer.hpp"

namespace {

template <typename Dtype>
__global__ void transpose_gpu(const int nthreads, const Dtype* from_data, Dtype* to_data, 
	const int* from_counts, const int* to_counts, const int* map, const int num_axes, int* buf) {
  CUDA_KERNEL_LOOP(index, nthreads) {
  	int* from_inds=buf + index * num_axes;

  	int from_index = index, to_index = 0;
  	for(int i = 0; i < num_axes; i++) {
		from_inds[i] = from_index / from_counts[i];
		from_index = from_index % from_counts[i];
	}
	for(int i = 0; i < num_axes; i++) {
		to_index += from_inds[map[i]] * to_counts[i];
	}

	*(to_data + to_index) = *(from_data + index);
  }
}

template <typename Dtype>
void TransposeLayer<Dtype>::Forward(GPUContext* context, const vector<Blob<Dtype>*>& bottom, 
		const vector<Blob<Dtype>*>& top) {
	const int nthreads = bottom[0]->count();

	transpose_gpu<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
        nthreads, bottom[0]->data<Context>(), top[0]->mutable_data<Context>(), 
        bottom_counts_.data<Context>(), top_counts_.data<Context>(), forward_map_.data<Context>(), 
          (int)bottom[0]->shape().size(), buf_.mutable_data<Context>());
}

template <typename Dtype>
void TransposeLayer<Dtype>::Backward(GPUContext* context, const vector<Blob<Dtype>*>& top,
    	const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
	if (!top[0]->propagate_down_) {
		return;
	}
	const int nthreads = bottom[0]->count();

	transpose_gpu<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
        nthreads, top[0]->gpu_diff(), bottom[0]->mutable_gpu_diff(), 
        top_counts_.data<Context>(), bottom_counts_.data<Context>(), backward_map_.data<Context>(), 
        (int)bottom[0]->shape().size(), buf_.mutable_data<Context>());
}

INSTANTIATE_LAYER_GPU_FUNCS(TransposeLayer);

}  // namespace
